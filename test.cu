#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

int main()
{
    hipDeviceProp_t deviceProp;
    int deviceCount;
    hipError_t hipError_t;
    hipError_t = hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        hipError_t = hipGetDeviceProperties(&deviceProp, i);

        cout << "设备 " << i + 1 << " 的主要属性： " << endl;
        cout << "设备显卡型号： " << deviceProp.name << endl;
        cout << "设备全局内存总量（以MB为单位）： " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;
        cout << "设备上一个线程块（Block）中可用的最大共享内存（以KB为单位）： " << deviceProp.sharedMemPerBlock / 1024
             << endl;
        cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << deviceProp.regsPerBlock << endl;
        cout << "设备上一个线程块（Block）可包含的最大线程数量： " << deviceProp.maxThreadsPerBlock << endl;
        cout << "设备的计算功能集（Compute Capability）的版本号： " << deviceProp.major << "." << deviceProp.minor
             << endl;
        cout << "设备上多处理器的数量： " << deviceProp.multiProcessorCount << endl;
    }
    getchar();
    return 0;
}